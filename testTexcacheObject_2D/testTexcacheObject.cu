#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define WIDTH 6
#define HEIGHT 2

#define CHECK_CUDART(x) do { \
  hipError_t res = (x); \
  if(res != hipSuccess) { \
    fprintf(stderr, "CUDART: %s = %d (%s) at (%s:%d)\n", #x, res, hipGetErrorString(res),__FILE__,__LINE__); \
    exit(1); \
  } \
} while(0) 

__global__ void printGpu_tex(hipTextureObject_t tex) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    if(tidx < WIDTH && tidy < HEIGHT){
        float x = tex2D<float>(tex, tidy, tidx);
        //float x = tex2D<float>(tex, float(tidx)+0.5, float(tidy)+0.5);
        printf("tex2D<float>(tex, %d, %d) = %f \n", tidy, tidx, x);
    }
}

__global__ void printGpu_vanilla(float* d_buffer, int pitch) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    if(tidx < WIDTH && tidy < HEIGHT){
        float x = d_buffer[tidy*pitch + tidx];
        printf("d_buffer[%d][%d] = %f \n", tidy, tidx, x);
    }
}

int main() {
    int width = WIDTH;
    int height = HEIGHT; 
    float h_buffer[12] = {1,2,3,4,5,6,7,8,9,10,11,12};

    float* d_buffer;
    size_t pitch;
    CHECK_CUDART(hipMallocPitch(&d_buffer, &pitch, sizeof(float)*width, height));
    CHECK_CUDART(hipMemset2D(d_buffer, pitch, 0, pitch, height));
    CHECK_CUDART(hipMemcpy2D(d_buffer, pitch, &h_buffer, sizeof(float)*width, sizeof(float)*width, height, hipMemcpyHostToDevice));
    printf("pitch = %d \n", pitch);

    //CUDA 5 texture objects: https://developer.nvidia.com/content/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = d_buffer;
    resDesc.res.pitch2D.pitchInBytes =  pitch;
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.desc.f = hipChannelFormatKindFloat;
    resDesc.res.pitch2D.desc.x = 32; // bits per channel
    resDesc.res.pitch2D.desc.y = 32;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex;
    CHECK_CUDART(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    dim3 grid(1, 1, 1); //assume one small block
    dim3 block(WIDTH, HEIGHT, 1);
    printGpu_tex<<<grid, block>>>(tex);
    CHECK_CUDART(hipGetLastError());
    printGpu_vanilla<<<grid, block>>>(d_buffer, pitch/sizeof(float));
    CHECK_CUDART(hipGetLastError());
    hipDestroyTextureObject(tex);
    hipFree(d_buffer);
}


