//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

#define DPM_DATA

void cufftForward_dpmData(){
    //int NbFeatures = 32;
    //int maxRows = 256;
    //int maxCols = 256;
    int NbFeatures = 1;
    int maxRows = 4;
    int maxCols = 4;

    int n[2] = {maxRows, maxCols};
    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {maxRows, maxCols}
                  0, //inembed
                  NbFeatures, //istride
                  1, //idist
                  0, //onembed
                  NbFeatures, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  NbFeatures /*batch*/));


#ifdef DPM_DATA
    float* h_in = readCsv_1dFloat(maxRows*maxCols*NbFeatures, "../plane_filter_0.csv");
#else
    float* h_in = (float*)malloc(sizeof(float) * maxRows*maxCols*NbFeatures);
    for(int i=0; i<(maxRows*maxCols*NbFeatures); i++){
        h_in[i] = (float)i; //* rand();
        printf("h_in[%d] = %f \n", i, h_in[i]);
    }
#endif

    hipfftComplex* h_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures);
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*maxRows*maxCols*NbFeatures));
    CHECK_CUDART(hipMemcpy(d_in, h_in, sizeof(float)*maxRows*maxCols*NbFeatures, hipMemcpyHostToDevice));
    CHECK_CUDART(hipMalloc(&d_freq, sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures));
    CHECK_CUDART(hipMemset(d_freq, 0, sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures));

    CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));

    CHECK_CUDART(hipMemcpy(h_freq, d_freq, sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures, hipMemcpyDeviceToHost));

    for(int i=0; i<(maxRows*maxCols*NbFeatures); i++){ 
        printf("cufft h_freq[%d].(x,y) = %0.10f,%0.10f \n", i, h_freq[i].x, h_freq[i].y);
    }
}

void deviceStuff(){
//    hipSetDevice(2); 
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
