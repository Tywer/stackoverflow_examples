#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

void cufftForward_dpmData(){
    int howMany = 2;
    int nRows = 4;
    int nCols = 4;

    int n[2] = {nRows, nCols};
    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {nRows, nCols}
                  0, //inembed
                  howMany, //istride
                  1, //idist
                  0, //onembed
                  howMany, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  howMany /*batch*/));

    float* h_in = (float*)malloc(sizeof(float) * nRows*nCols*howMany);
    for(int i=0; i<(nRows*nCols*howMany); i++){
        h_in[i] = (float)i;
        printf("h_in[%d] = %f \n", i, h_in[i]);
    }

    hipfftComplex* h_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*nRows*nCols*howMany);
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*nRows*nCols*howMany));
    CHECK_CUDART(hipMemcpy(d_in, h_in, sizeof(float)*nRows*nCols*howMany, hipMemcpyHostToDevice));
    CHECK_CUDART(hipMalloc(&d_freq, sizeof(hipfftComplex)*nRows*nCols*howMany));
    CHECK_CUDART(hipMemset(d_freq, 0, sizeof(hipfftComplex)*nRows*nCols*howMany));

    CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));

    CHECK_CUDART(hipMemcpy(h_freq, d_freq, sizeof(hipfftComplex)*nRows*nCols*howMany, hipMemcpyDeviceToHost));
    for(int i=0; i<(nRows*nCols*howMany); i++){ 
        printf("cufft h_freq[%d].(x,y) = %f,%f \n", i, h_freq[i].x, h_freq[i].y);
    }
}

void deviceStuff(){
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
