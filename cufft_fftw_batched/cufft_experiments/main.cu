#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

void cufftForward_dpmData(){
    int howMany = 1;
    int maxRows = 4;
    int maxCols = 4;

    int n[2] = {maxRows, maxCols};
    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {maxRows, maxCols}
                  0, //inembed
                  howMany, //istride
                  1, //idist
                  0, //onembed
                  howMany, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  howMany /*batch*/));

    float* h_in = (float*)malloc(sizeof(float) * maxRows*maxCols*howMany);
    for(int i=0; i<(maxRows*maxCols*howMany); i++){
        h_in[i] = (float)i; //* rand();
        printf("h_in[%d] = %f \n", i, h_in[i]);
    }

    hipfftComplex* h_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*maxRows*maxCols*howMany);
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*maxRows*maxCols*howMany));
    CHECK_CUDART(hipMemcpy(d_in, h_in, sizeof(float)*maxRows*maxCols*howMany, hipMemcpyHostToDevice));
    CHECK_CUDART(hipMalloc(&d_freq, sizeof(hipfftComplex)*maxRows*maxCols*howMany));
    CHECK_CUDART(hipMemset(d_freq, 0, sizeof(hipfftComplex)*maxRows*maxCols*howMany));

    CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));

    CHECK_CUDART(hipMemcpy(h_freq, d_freq, sizeof(hipfftComplex)*maxRows*maxCols*howMany, hipMemcpyDeviceToHost));

    for(int i=0; i<(maxRows*maxCols*howMany); i++){ 
        printf("cufft h_freq[%d].(x,y) = %f,%f \n", i, h_freq[i].x, h_freq[i].y);
    }
}

void deviceStuff(){
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
