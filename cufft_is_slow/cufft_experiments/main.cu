//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

#define OUT_OF_PLACE
void cufftForward_experiment(){
    int depth = 32;
    int nRows = 256;
    int nCols = 256;
    int nIter = 8;
    int n[2] = {nRows, nCols};

    #ifdef OUT_OF_PLACE
    //if nCols is even, cols_padded = (nCols+2). if nCols is odd, cols_padded = (nCols+1)
    int cols_padded = 2*(nCols/2 + 1); //allocate this width, but tell FFTW that it's nCols width
    int inembed[2] = {nRows, 2*(nCols/2 + 1)};
    int onembed[2] = {nRows, (nCols/2 + 1)}; //default -- equivalent ot onembed=NULL in FFTW
    #else
    int cols_padded = nCols;
    int inembed[2] = {nRows, nCols};
    int onembed[2] = {nRows, (nCols/2 + 1)}; //default -- equivalent of onembed=NULL
    #endif

    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {nRows, nCols}
                  inembed, //inembed
                  depth, //istride
                  1, //idist
                  onembed, //onembed
                  depth, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  depth /*batch*/));
    
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*nRows*cols_padded*depth)); 
    #ifdef OUT_OF_PLACE
    d_freq = reinterpret_cast<hipfftComplex*>(d_in);
    #else
    CHECK_CUDART(hipMalloc(&d_freq, sizeof(hipfftComplex)*nRows*cols_padded*depth)); 
    #endif    

    double start = read_timer();
    for(int i=0; i<nIter; i++){
        CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));
    }
    CHECK_CUDART(hipDeviceSynchronize());
    double responseTime = read_timer() - start;
    printf("did %d FFT calls in %f ms \n", nIter, responseTime);

    //TODO: free memory
}

void deviceStuff(){
//    hipSetDevice(2); 
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_experiment();
    return 0;
}
