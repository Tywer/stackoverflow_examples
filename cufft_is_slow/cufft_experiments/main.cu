//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

void cufftForward_dpmData(){
    int depth = 32;
    int maxRows = 1024;
    int maxCols = 1024;
    int nIter = 2;

    int n[2] = {maxRows, maxCols};
    int cols_padded;
    //if maxCols is even, cols_padded = (maxCols+2). if maxCols is odd, cols_padded = (maxCols+1)
    cols_padded = 2*(maxCols/2 + 1); //allocate this width, but tell FFTW that it's maxCols width
    int inembed[2] = {maxRows, 2*(maxCols/2 + 1)};
    int onembed[2] = {maxRows, (maxCols/2 + 1)}; //default -- equivalent ot onembed=NULL in FFTW

    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {maxRows, maxCols}
                  inembed, //inembed
                  depth, //istride
                  1, //idist
                  onembed, //onembed
                  depth, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  depth /*batch*/));
    
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*maxRows*cols_padded*depth)); //cols_padded varies depending on whether in-place or not
    d_freq = reinterpret_cast<hipfftComplex*>(d_in);
    
    double start = read_timer();
    for(int i=0; i<nIter; i++){

        CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));
    }
    CHECK_CUDART(hipDeviceSynchronize());
    double responseTime = read_timer() - start;
    printf("did %d FFT calls in %f ms \n", nIter, responseTime);

    //TODO: free memory
}

void deviceStuff(){
//    hipSetDevice(2); 
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
