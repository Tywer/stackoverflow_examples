//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

void cufftForward_dpmData(){
    int NbFeatures = 32;
    int maxRows = 256;
    int maxCols = 256;
    int nIter = 10;

    int n[2] = {maxRows, maxCols};
    int cols_padded;
    //if maxCols is even, cols_padded = (maxCols+2). if maxCols is odd, cols_padded = (maxCols+1)
    cols_padded = 2*(maxCols/2 + 1); //allocate this width, but tell FFTW that it's maxCols width
    int inembed[2] = {maxRows, 2*(maxCols/2 + 1)};
    int onembed[2] = {maxRows, (maxCols/2 + 1)}; //default -- equivalent ot onembed=NULL in FFTW

    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {maxRows, maxCols}
                  inembed, //inembed
                  NbFeatures, //istride
                  1, //idist
                  onembed, //onembed
                  NbFeatures, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  NbFeatures /*batch*/));
    //CHECK_CUFFT(cufftSetCompatibilityMode(forwardPlan, CUFFT_COMPATIBILITY_FFTW_ALL));
    
    float* h_in = (float*)malloc(sizeof(float)*maxRows*cols_padded*NbFeatures); //note cols_padded instead of maxCols
    memset(h_in, 0, sizeof(float)*maxRows*cols_padded*NbFeatures);


#if 0
    for(int row=0; row<maxRows; row++){
        for(int col=0; col<maxCols; col++){ //iterate through maxCols, but multiply row by cols_padded.
            for(int depth=0; depth<NbFeatures; depth++){
                int idx = row * cols_padded * NbFeatures +
                          col * NbFeatures +
                          depth;
                h_in[idx] = row * maxCols * NbFeatures + //using maxCols instead of cols_padded here, so that input data is same for in-place and out-of-place versions
                            col * NbFeatures +
                            depth;
//                printf("h_in[row=%d, col=%d, depth=%d] = %f \n", row, col, depth, h_in[idx]);
            }
        }
    }
#endif
    hipfftComplex* h_freq = reinterpret_cast<hipfftComplex*>(h_in); 
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*maxRows*cols_padded*NbFeatures)); //cols_padded varies depending on whether in-place or not
//    CHECK_CUDART(hipMemcpy(d_in, h_in, sizeof(float)*maxRows*cols_padded*NbFeatures, hipMemcpyHostToDevice));
    d_freq = reinterpret_cast<hipfftComplex*>(d_in);
    
    double start = read_timer();
    for(int i=0; i<nIter; i++){

        CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));
        CHECK_CUDART(hipDeviceSynchronize());
    }
    double responseTime = read_timer() - start;
    printf("did %d FFT calls in %f ms \n", nIter, responseTime);

//    CHECK_CUDART(hipMemcpy(h_freq, d_freq, sizeof(float)*maxRows*cols_padded*NbFeatures, hipMemcpyDeviceToHost)); //TODO: copy exactly the right amount of space to host

    for(int i=0; i<(maxRows*maxCols*NbFeatures); i++){
//         printf("    cufft h_freq[%d].(x,y) = %0.0f, %0.0f \n", i, h_freq[i].x, h_freq[i].y); //for StackOverflow 
        //printf("cufft h_freq[%d].(x,y) = %0.10f,%0.10f \n", i, h_freq[i].x, h_freq[i].y);
    }

    //TODO: free memory
}

void deviceStuff(){
//    hipSetDevice(2); 
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
