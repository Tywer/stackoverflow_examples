//#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

//#define DPM_DATA
//#define INPLACE

void cufftForward_dpmData(){
    int NbFeatures = 32;
    int maxRows = 256;
    int maxCols = 256;
    //int NbFeatures = 2;
    //int maxRows = 4;
    //int maxCols = 4;

    int n[2] = {maxRows, maxCols};
    int cols_padded;
#ifdef INPLACE
    //if maxCols is even, cols_padded = (maxCols+2). if maxCols is odd, cols_padded = (maxCols+1)
    cols_padded = 2*(maxCols/2 + 1); //allocate this width, but tell FFTW that it's maxCols width
    int inembed[2] = {maxRows, 2*(maxCols/2 + 1)};
    int onembed[2] = {maxRows, (maxCols/2 + 1)}; //default -- equivalent ot onembed=NULL in FFTW
#else
    cols_padded = maxCols;
    int inembed[2] = {maxRows, maxCols};
    int onembed[2] = {maxRows, (maxCols/2 + 1)}; //default -- equivalent ot onembed=NULL in FFTW
#endif

    hipfftHandle forwardPlan; 
    float* d_in; hipfftComplex* d_freq; 

    CHECK_CUFFT(hipfftPlanMany(&forwardPlan,
                  2, //rank
                  n, //dimensions = {maxRows, maxCols}
                  inembed, //inembed
                  NbFeatures, //istride
                  1, //idist
                  onembed, //onembed
                  NbFeatures, //ostride
                  1, //odist
                  HIPFFT_R2C, //hipfftType
                  NbFeatures /*batch*/));
    //CHECK_CUFFT(cufftSetCompatibilityMode(forwardPlan, CUFFT_COMPATIBILITY_FFTW_ALL));
    
    float* h_in = (float*)malloc(sizeof(float)*maxRows*cols_padded*NbFeatures); //note cols_padded instead of maxCols
    memset(h_in, 0, sizeof(float)*maxRows*cols_padded*NbFeatures);

#ifdef DPM_DATA
    free(h_in);
    h_in = readCsv_1dFloat(maxRows*maxCols*NbFeatures, "../plane_filter_0.csv"); //TODO: make this code understand the padding.
#else
    for(int row=0; row<maxRows; row++){
        for(int col=0; col<maxCols; col++){ //iterate through maxCols, but multiply row by cols_padded.
            for(int depth=0; depth<NbFeatures; depth++){
                int idx = row * cols_padded * NbFeatures +
                          col * NbFeatures +
                          depth;
                h_in[idx] = row * maxCols * NbFeatures + //using maxCols instead of cols_padded here, so that input data is same for in-place and out-of-place versions
                            col * NbFeatures +
                            depth;
//                printf("h_in[row=%d, col=%d, depth=%d] = %f \n", row, col, depth, h_in[idx]);
            }
        }
    }
#endif

#ifdef INPLACE
    hipfftComplex* h_freq = reinterpret_cast<hipfftComplex*>(h_in); 
#else
    hipfftComplex* h_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*maxRows*cols_padded*NbFeatures); //TODO: perhaps allocate smaller space
#endif
    CHECK_CUDART(hipMalloc(&d_in, sizeof(float)*maxRows*cols_padded*NbFeatures)); //cols_padded varies depending on whether in-place or not
    CHECK_CUDART(hipMemcpy(d_in, h_in, sizeof(float)*maxRows*cols_padded*NbFeatures, hipMemcpyHostToDevice));

#ifdef INPLACE
    d_freq = reinterpret_cast<hipfftComplex*>(d_in);
#else
    CHECK_CUDART(hipMalloc(&d_freq, sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures));
    CHECK_CUDART(hipMemset(d_freq, 0, sizeof(hipfftComplex)*maxRows*maxCols*NbFeatures));
#endif
    
    CHECK_CUFFT(hipfftExecR2C(forwardPlan, d_in, d_freq));

    //this is fine for in-place. for out-of-place, we need cols_padded = 2*(maxCols/2 + 1) here too, to get enough data out.
    CHECK_CUDART(hipMemcpy(h_freq, d_freq, sizeof(float)*maxRows*cols_padded*NbFeatures, hipMemcpyDeviceToHost)); //TODO: copy exactly the right amount of space to host

    for(int i=0; i<(maxRows*maxCols*NbFeatures); i++){
//         printf("    cufft h_freq[%d].(x,y) = %0.0f, %0.0f \n", i, h_freq[i].x, h_freq[i].y); //for StackOverflow 
        //printf("cufft h_freq[%d].(x,y) = %0.10f,%0.10f \n", i, h_freq[i].x, h_freq[i].y);
    }

    //TODO: free memory
}

void deviceStuff(){
//    hipSetDevice(2); 
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize());
    cufftForward_dpmData();
    return 0;
}
