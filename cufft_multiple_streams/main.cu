#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hipfft/hipfft.h>
#include "helpers.h"
using namespace std;

void cufftForward_streams_experiment(){

    //note: in my 'real' app, I'm already using batch mode. 
    // I need to do multiple batch mode operations, each of which operates on a different chunk of data. 
    // Hence, the need for doing cuFFT calls in streams
    //
    // Also, in my real application, I overlap memcpys and FFTs. 
    // But, for the sake of this exercise, I'm just studying the overlap among FFTs. (or lack of overlap, in this case)

    int nRows = 16; //each FFT is tiny; doesn't saturate the GPU
    int nCols = 16;
    int number_of_FFTs = 3;

    vector<hipfftHandle> forwardPlan(number_of_FFTs);
    vector<float*> d_in(number_of_FFTs);
    vector<hipfftComplex*> d_freq(number_of_FFTs);

    int nStreams = number_of_FFTs;
    vector<hipStream_t> streams(nStreams);
    for(int s=0; s<nStreams; s++){
        CHECK_CUDART(hipStreamCreate(&streams[s]));
    }

    for(int i=0; i<number_of_FFTs; i++){
        CHECK_CUFFT(hipfftPlan2d(&forwardPlan[i], nCols, nRows, HIPFFT_R2C));
        CHECK_CUFFT(hipfftSetStream(forwardPlan[i], streams[i]));

        CHECK_CUDART(hipMalloc(&d_in[i], sizeof(float)*nRows*nCols));
        CHECK_CUDART(hipMemset(d_in[i], 0, sizeof(float)*nRows*nCols));
        d_freq[i] = reinterpret_cast<hipfftComplex *>(d_in[i]);
    }
    CHECK_CUDART(hipDeviceSynchronize()); 
   
    double start = read_timer();
    for(int i=0; i<number_of_FFTs; i++){
        CHECK_CUFFT(hipfftExecR2C(forwardPlan[i], d_in[i], d_freq[i]));
    }
    double forwardTime = read_timer() - start;
    printf("time for %d forward FFTs in streams = %f \n", number_of_FFTs, forwardTime);
    printf("avg time per FFT = %f \n", forwardTime/number_of_FFTs);
}

void deviceStuff(){
    int device; hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("deviceName = %s \n", prop.name);
}

int main (int argc, char **argv){
    deviceStuff();
    CHECK_CUDART(hipDeviceSynchronize()); //warmup
    cufftForward_streams_experiment();
    return 0;
}
